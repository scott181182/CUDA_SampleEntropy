#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include "./book.h"
#include "./sampen_gpu.h"



#define KERNEL_SIZE 1024



__device__ double chebyshev_distance_gpu(double* A, double* B, unsigned int length) {
    double d = 0;
    for(unsigned int i = 0; i < length; i++) {
        d = max(abs(A[i] - B[i]), d);
    }
    return d;
}

__global__ void sampen_kernel(double* data, unsigned int length, unsigned int m, double r, unsigned int *AB) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if(i > length - m) { return; }
    if(i + m < length) {
        // Start at -1 to account for the self-match.
        int A = -1, B = -1;
        for(unsigned int j = 0; j <= length - m; j++) {
            if(chebyshev_distance_gpu(&data[i], &data[j], m) < r) {
                B++;
                if(j + m < length && abs(data[i + m] - data[j + m]) < r) {
                    A++;
                }
            }
        }
        AB[2 * i] = A;
        AB[2 * i + 1] = B;
    } else {
        int B = -1;
        for(unsigned int j = 0; j <= length - m; j++) {
            if(chebyshev_distance_gpu(&data[i], &data[j], m) < r) { B++; }
        }
        AB[2 * i] = 0;
        AB[2 * i + 1] = B;
    }
}
double sampen_gpu(double* data, unsigned int length, unsigned int m, double r) {
    if(m < 1) { return 0; }

    double *data_dev;
    HANDLE_ERROR(hipMalloc((void**)&data_dev, sizeof(double) * length));
    HANDLE_ERROR(hipMemcpy(data_dev, data, sizeof(double) * length, hipMemcpyHostToDevice));

    /** Interleaved array of window-wise {A, B} pairings. */
    unsigned int AB_length = 2 * (length - m + 1);
    size_t AB_pitch = AB_length * sizeof(unsigned int);
    unsigned int *AB_dev;
    HANDLE_ERROR(hipMalloc((void**)&AB_dev, AB_pitch));

    const unsigned int BLOCK_SIZE = (length / KERNEL_SIZE) + 1;
    sampen_kernel<<<KERNEL_SIZE, BLOCK_SIZE>>>(data_dev, length, m, r, AB_dev);

    hipFree(data_dev);
    unsigned int *AB_arr = (unsigned int*)malloc(AB_pitch);
    HANDLE_ERROR(hipMemcpy(AB_arr, AB_dev, AB_pitch, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipFree(AB_dev));

    unsigned long A = 0, B = 0;
    for(unsigned int i = 0; i < AB_length; i += 2) {
        A += AB_arr[i];
        B += AB_arr[i + 1];
    }
    free(AB_arr);

    printf("A=%ld, B=%ld\n", A, B);
    printf("A/B=%16.16lf\n", (double)A / (double)B);
    return -log((double)A / (double)B);
}
