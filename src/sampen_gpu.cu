#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include "./book.h"



#define KERNEL_SIZE 1024



__device__ float chebyshev_distance_gpu(float* A, float* B, unsigned int length) {
    float d = 0;
    for(unsigned int i = 0; i < length; i++) {
        d = max(abs(A[i] - B[i]), d);
    }
    return d;
}

__global__ void sampen_kernel(float* data, unsigned int length, unsigned int m, float r, unsigned int *AB) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if(i > length - m) { return; }
    if(i + m < length) {
        // Start at -1 to account for the self-match.
        int A = -1, B = -1;
        for(unsigned int j = 0; j <= length - m; j++) {
            if(chebyshev_distance_gpu(&data[i], &data[j], m) < r) {
                B++;
                if(j + m < length && abs(data[i + m] - data[j + m]) < r) {
                    A++;
                }
            }
        }
        AB[2 * i] = A;
        AB[2 * i + 1] = B;
    } else {
        int B = -1;
        for(unsigned int j = 0; j <= length - m; j++) {
            if(chebyshev_distance_gpu(&data[i], &data[j], m) < r) { B++; }
        }
        AB[2 * i] = 0;
        AB[2 * i + 1] = B;
    }
}
float sampen_gpu(float* data, unsigned int length, unsigned int m, float r) {
    if(m < 1) { return 0; }

    float *data_dev;
    HANDLE_ERROR(hipMalloc((void**)&data_dev, sizeof(float) * length));
    HANDLE_ERROR(hipMemcpy(data_dev, data, sizeof(float) * length, hipMemcpyHostToDevice));

    /** Interleaved array of window-wise {A, B} pairings. */
    unsigned int AB_length = 2 * (length - m + 1);
    size_t AB_pitch = AB_length * sizeof(unsigned int);
    unsigned int *AB_dev;
    HANDLE_ERROR(hipMalloc((void**)&AB_dev, AB_pitch));

    const unsigned int BLOCK_SIZE = (length / KERNEL_SIZE) + 1;
    sampen_kernel<<<KERNEL_SIZE, BLOCK_SIZE>>>(data_dev, length, m, r, AB_dev);

    hipFree(data_dev);
    unsigned int *AB_arr = (unsigned int*)malloc(AB_pitch);
    HANDLE_ERROR(hipMemcpy(AB_arr, AB_dev, AB_pitch, hipMemcpyDeviceToHost));
    HANDLE_ERROR(hipFree(AB_dev));

    unsigned long A = 0, B = 0;
    for(unsigned int i = 0; i < AB_length; i += 2) {
        A += AB_arr[i];
        B += AB_arr[i + 1];
    }
    free(AB_arr);

    printf("A=%ld, B=%ld\n", A, B);
    printf("A/B=%16.16lf\n", (float)A / (float)B);
    return -log((float)A / (float)B);
}
